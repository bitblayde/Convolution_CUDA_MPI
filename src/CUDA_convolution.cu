#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <ctime>

#include "convolucion_CUDA.cuh"


__global__
void convolution( int w, int h, int *src, int *dst){ //const CImg<float> &img){
    int id_thread = threadIdx.x + blockDim.x * blockIdx.x;

    unsigned int resultado, current_p;

    if (id_thread < w - 2){

      int kernel[5][5] = {
     {1, 4, 7, 4, 1},
     {4, 16, 26, 16, 4},
     {7, 26, 41, 26, 7},
     {4, 16, 26, 16, 4},
     {1, 4, 7, 4, 1}
     };


      for(int k = 2; k <= h - 2; k++){

        resultado = 0.0f;

        for(int i = -2; i <= 2; i++){
          for(int j = -2; j <= 2; j++){

            if( (k + j >= h) || ( i + id_thread >= w ) ){
              current_p = *(src + id_thread * h + k);
            }
            else{
              current_p = *( src + (id_thread + i) * h + (k + j) );
            }

            resultado = resultado + current_p * kernel[i+2][j+2];
          }
        }
        *(dst + id_thread * h + k) = resultado/273;
      }

    }

}

double convolution_interface( int w, int h, int *src){
  clock_t begin_computo, end_computo;

  dim3 block_number((w/32) + 1, 1, 1);
  dim3 block_dim(32, 1, 1);
  int *__convolution_pointer = nullptr;
  int size = sizeof(int)*(w*h);

  int *dst = nullptr;

  hipMalloc((void **) &__convolution_pointer, size);
  hipMalloc((void **) &dst, size);

  begin_computo = clock();

  hipMemcpy(__convolution_pointer, src, size, hipMemcpyHostToDevice);
  convolution<<< block_number, block_dim >>>(w, h, __convolution_pointer, dst);
  hipDeviceSynchronize();
  hipMemcpy(src, dst, size, hipMemcpyDeviceToHost);

  end_computo = clock();

  hipFree(__convolution_pointer);
  hipFree(dst);

  return double(end_computo - begin_computo) / CLOCKS_PER_SEC;
}
